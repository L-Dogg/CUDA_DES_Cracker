#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <cstdint>
#include <ctime>

#define FIRSTBIT	0x8000000000000000
#define BLOCK_SIZE	1024
#define BLOCKS		2048
#define KNOWN_ZEROS	37
#define MSGLEN		1

__device__ int work = 1;

// Host matrices:
const int PC1[56] = {
	57, 49, 41, 33, 25, 17,  9,
	1, 58, 50, 42, 34, 26, 18,
	10,  2, 59, 51, 43, 35, 27,
	19, 11,  3, 60, 52, 44, 36,
	63, 55, 47, 39, 31, 23, 15,
	7, 62, 54, 46, 38, 30, 22,
	14,  6, 61, 53, 45, 37, 29,
	21, 13,  5, 28, 20, 12,  4
};
const int Rotations[16] = {
	1, 1, 2, 2, 2, 2, 2, 2, 1, 2, 2, 2, 2, 2, 2, 1
};
const int PC2[48] = {
	14, 17, 11, 24,  1,  5,
	3, 28, 15,  6, 21, 10,
	23, 19, 12,  4, 26,  8,
	16,  7, 27, 20, 13,  2,
	41, 52, 31, 37, 47, 55,
	30, 40, 51, 45, 33, 48,
	44, 49, 39, 56, 34, 53,
	46, 42, 50, 36, 29, 32
};
const int InitialPermutation[64] = {
	58, 50, 42, 34, 26, 18, 10,  2,
	60, 52, 44, 36, 28, 20, 12,  4,
	62, 54, 46, 38, 30, 22, 14,  6,
	64, 56, 48, 40, 32, 24, 16,  8,
	57, 49, 41, 33, 25, 17,  9,  1,
	59, 51, 43, 35, 27, 19, 11,  3,
	61, 53, 45, 37, 29, 21, 13,  5,
	63, 55, 47, 39, 31, 23, 15,  7
};
const int FinalPermutation[64] = {
	40,  8, 48, 16, 56, 24, 64, 32,
	39,  7, 47, 15, 55, 23, 63, 31,
	38,  6, 46, 14, 54, 22, 62, 30,
	37,  5, 45, 13, 53, 21, 61, 29,
	36,  4, 44, 12, 52, 20, 60, 28,
	35,  3, 43, 11, 51, 19, 59, 27,
	34,  2, 42, 10, 50, 18, 58, 26,
	33,  1, 41,  9, 49, 17, 57, 25
};
const int DesExpansion[48] = {
	32,  1,  2,  3,  4,  5,  4,  5,
	6,  7,  8,  9,  8,  9, 10, 11,
	12, 13, 12, 13, 14, 15, 16, 17,
	16, 17, 18, 19, 20, 21, 20, 21,
	22, 23, 24, 25, 24, 25, 26, 27,
	28, 29, 28, 29, 30, 31, 32,  1
};
const int DesSbox[8][4][16] = {
	{
		{ 14,  4, 13,  1,  2, 15, 11,  8,  3, 10,  6, 12,  5,  9,  0,  7 },
		{ 0, 15,  7,  4, 14,  2, 13,  1, 10,  6, 12, 11,  9,  5,  3,  8 },
		{ 4,  1, 14,  8, 13,  6,  2, 11, 15, 12,  9,  7,  3, 10,  5,  0 },
		{ 15, 12,  8,  2,  4,  9,  1,  7,  5, 11,  3, 14, 10,  0,  6, 13 },
	},

	{
		{ 15,  1,  8, 14,  6, 11,  3,  4,  9,  7,  2, 13, 12,  0,  5, 10 },
		{ 3, 13,  4,  7, 15,  2,  8, 14, 12,  0,  1, 10,  6,  9, 11,  5 },
		{ 0, 14,  7, 11, 10,  4, 13,  1,  5,  8, 12,  6,  9,  3,  2, 15 },
		{ 13,  8, 10,  1,  3, 15,  4,  2, 11,  6,  7, 12,  0,  5, 14,  9 },
	},

	{
		{ 10,  0,  9, 14,  6,  3, 15,  5,  1, 13, 12,  7, 11,  4,  2,  8 },
		{ 13,  7,  0,  9,  3,  4,  6, 10,  2,  8,  5, 14, 12, 11, 15,  1 },
		{ 13,  6,  4,  9,  8, 15,  3,  0, 11,  1,  2, 12,  5, 10, 14,  7 },
		{ 1, 10, 13,  0,  6,  9,  8,  7,  4, 15, 14,  3, 11,  5,  2, 12 },
	},

	{
		{ 7, 13, 14,  3,  0,  6,  9, 10,  1,  2,  8,  5, 11, 12,  4, 15 },
		{ 13,  8, 11,  5,  6, 15,  0,  3,  4,  7,  2, 12,  1, 10, 14,  9 },
		{ 10,  6,  9,  0, 12, 11,  7, 13, 15,  1,  3, 14,  5,  2,  8,  4 },
		{ 3, 15,  0,  6, 10,  1, 13,  8,  9,  4,  5, 11, 12,  7,  2, 14 },
	},

	{
		{ 2, 12,  4,  1,  7, 10, 11,  6,  8,  5,  3, 15, 13,  0, 14,  9 },
		{ 14, 11,  2, 12,  4,  7, 13,  1,  5,  0, 15, 10,  3,  9,  8,  6 },
		{ 4,  2,  1, 11, 10, 13,  7,  8, 15,  9, 12,  5,  6,  3,  0, 14 },
		{ 11,  8, 12,  7,  1, 14,  2, 13,  6, 15,  0,  9, 10,  4,  5,  3 },
	},

	{
		{ 12,  1, 10, 15,  9,  2,  6,  8,  0, 13,  3,  4, 14,  7,  5, 11 },
		{ 10, 15,  4,  2,  7, 12,  9,  5,  6,  1, 13, 14,  0, 11,  3,  8 },
		{ 9, 14, 15,  5,  2,  8, 12,  3,  7,  0,  4, 10,  1, 13, 11,  6 },
		{ 4,  3,  2, 12,  9,  5, 15, 10, 11, 14,  1,  7,  6,  0,  8, 13 },
	},

	{
		{ 4, 11,  2, 14, 15,  0,  8, 13,  3, 12,  9,  7,  5, 10,  6,  1 },
		{ 13,  0, 11,  7,  4,  9,  1, 10, 14,  3,  5, 12,  2, 15,  8,  6 },
		{ 1,  4, 11, 13, 12,  3,  7, 14, 10, 15,  6,  8,  0,  5,  9,  2 },
		{ 6, 11, 13,  8,  1,  4, 10,  7,  9,  5,  0, 15, 14,  2,  3, 12 },
	},

	{
		{ 13,  2,  8,  4,  6, 15, 11,  1, 10,  9,  3, 14,  5,  0, 12,  7 },
		{ 1, 15, 13,  8, 10,  3,  7,  4, 12,  5,  6, 11,  0, 14,  9,  2 },
		{ 7, 11,  4,  1,  9, 12, 14,  2,  0,  6, 10, 13, 15,  3,  5,  8 },
		{ 2,  1, 14,  7,  4, 10,  8, 13, 15, 12,  9,  0,  3,  5,  6, 11 },
	},
};
const int Pbox[32] = {
	16,  7, 20, 21, 29, 12, 28, 17,
	1, 15, 23, 26,  5, 18, 31, 10,
	2,  8, 24, 14, 32, 27,  3,  9,
	19, 13, 30,  6, 22, 11,  4, 25
};

// Device matrices:
__constant__ int d_PC1[56] = {
	57, 49, 41, 33, 25, 17,  9,
	1, 58, 50, 42, 34, 26, 18,
	10,  2, 59, 51, 43, 35, 27,
	19, 11,  3, 60, 52, 44, 36,
	63, 55, 47, 39, 31, 23, 15,
	7, 62, 54, 46, 38, 30, 22,
	14,  6, 61, 53, 45, 37, 29,
	21, 13,  5, 28, 20, 12,  4
};
__constant__ int d_Rotations[16] = {
	1, 1, 2, 2, 2, 2, 2, 2, 1, 2, 2, 2, 2, 2, 2, 1
};
__constant__ int d_PC2[48] = {
	14, 17, 11, 24,  1,  5,
	3, 28, 15,  6, 21, 10,
	23, 19, 12,  4, 26,  8,
	16,  7, 27, 20, 13,  2,
	41, 52, 31, 37, 47, 55,
	30, 40, 51, 45, 33, 48,
	44, 49, 39, 56, 34, 53,
	46, 42, 50, 36, 29, 32
};
__constant__ int d_InitialPermutation[64] = {
	58, 50, 42, 34, 26, 18, 10,  2,
	60, 52, 44, 36, 28, 20, 12,  4,
	62, 54, 46, 38, 30, 22, 14,  6,
	64, 56, 48, 40, 32, 24, 16,  8,
	57, 49, 41, 33, 25, 17,  9,  1,
	59, 51, 43, 35, 27, 19, 11,  3,
	61, 53, 45, 37, 29, 21, 13,  5,
	63, 55, 47, 39, 31, 23, 15,  7
};
__constant__ int d_FinalPermutation[64] = {
	40,  8, 48, 16, 56, 24, 64, 32,
	39,  7, 47, 15, 55, 23, 63, 31,
	38,  6, 46, 14, 54, 22, 62, 30,
	37,  5, 45, 13, 53, 21, 61, 29,
	36,  4, 44, 12, 52, 20, 60, 28,
	35,  3, 43, 11, 51, 19, 59, 27,
	34,  2, 42, 10, 50, 18, 58, 26,
	33,  1, 41,  9, 49, 17, 57, 25
};
__constant__ int d_DesExpansion[48] = {
	32,  1,  2,  3,  4,  5,  4,  5,
	6,  7,  8,  9,  8,  9, 10, 11,
	12, 13, 12, 13, 14, 15, 16, 17,
	16, 17, 18, 19, 20, 21, 20, 21,
	22, 23, 24, 25, 24, 25, 26, 27,
	28, 29, 28, 29, 30, 31, 32,  1
};
__constant__ int d_DesSbox[8][4][16] = {
	{
		{ 14,  4, 13,  1,  2, 15, 11,  8,  3, 10,  6, 12,  5,  9,  0,  7 },
		{ 0, 15,  7,  4, 14,  2, 13,  1, 10,  6, 12, 11,  9,  5,  3,  8 },
		{ 4,  1, 14,  8, 13,  6,  2, 11, 15, 12,  9,  7,  3, 10,  5,  0 },
		{ 15, 12,  8,  2,  4,  9,  1,  7,  5, 11,  3, 14, 10,  0,  6, 13 },
	},

	{
		{ 15,  1,  8, 14,  6, 11,  3,  4,  9,  7,  2, 13, 12,  0,  5, 10 },
		{ 3, 13,  4,  7, 15,  2,  8, 14, 12,  0,  1, 10,  6,  9, 11,  5 },
		{ 0, 14,  7, 11, 10,  4, 13,  1,  5,  8, 12,  6,  9,  3,  2, 15 },
		{ 13,  8, 10,  1,  3, 15,  4,  2, 11,  6,  7, 12,  0,  5, 14,  9 },
	},

	{
		{ 10,  0,  9, 14,  6,  3, 15,  5,  1, 13, 12,  7, 11,  4,  2,  8 },
		{ 13,  7,  0,  9,  3,  4,  6, 10,  2,  8,  5, 14, 12, 11, 15,  1 },
		{ 13,  6,  4,  9,  8, 15,  3,  0, 11,  1,  2, 12,  5, 10, 14,  7 },
		{ 1, 10, 13,  0,  6,  9,  8,  7,  4, 15, 14,  3, 11,  5,  2, 12 },
	},

	{
		{ 7, 13, 14,  3,  0,  6,  9, 10,  1,  2,  8,  5, 11, 12,  4, 15 },
		{ 13,  8, 11,  5,  6, 15,  0,  3,  4,  7,  2, 12,  1, 10, 14,  9 },
		{ 10,  6,  9,  0, 12, 11,  7, 13, 15,  1,  3, 14,  5,  2,  8,  4 },
		{ 3, 15,  0,  6, 10,  1, 13,  8,  9,  4,  5, 11, 12,  7,  2, 14 },
	},

	{
		{ 2, 12,  4,  1,  7, 10, 11,  6,  8,  5,  3, 15, 13,  0, 14,  9 },
		{ 14, 11,  2, 12,  4,  7, 13,  1,  5,  0, 15, 10,  3,  9,  8,  6 },
		{ 4,  2,  1, 11, 10, 13,  7,  8, 15,  9, 12,  5,  6,  3,  0, 14 },
		{ 11,  8, 12,  7,  1, 14,  2, 13,  6, 15,  0,  9, 10,  4,  5,  3 },
	},

	{
		{ 12,  1, 10, 15,  9,  2,  6,  8,  0, 13,  3,  4, 14,  7,  5, 11 },
		{ 10, 15,  4,  2,  7, 12,  9,  5,  6,  1, 13, 14,  0, 11,  3,  8 },
		{ 9, 14, 15,  5,  2,  8, 12,  3,  7,  0,  4, 10,  1, 13, 11,  6 },
		{ 4,  3,  2, 12,  9,  5, 15, 10, 11, 14,  1,  7,  6,  0,  8, 13 },
	},

	{
		{ 4, 11,  2, 14, 15,  0,  8, 13,  3, 12,  9,  7,  5, 10,  6,  1 },
		{ 13,  0, 11,  7,  4,  9,  1, 10, 14,  3,  5, 12,  2, 15,  8,  6 },
		{ 1,  4, 11, 13, 12,  3,  7, 14, 10, 15,  6,  8,  0,  5,  9,  2 },
		{ 6, 11, 13,  8,  1,  4, 10,  7,  9,  5,  0, 15, 14,  2,  3, 12 },
	},

	{
		{ 13,  2,  8,  4,  6, 15, 11,  1, 10,  9,  3, 14,  5,  0, 12,  7 },
		{ 1, 15, 13,  8, 10,  3,  7,  4, 12,  5,  6, 11,  0, 14,  9,  2 },
		{ 7, 11,  4,  1,  9, 12, 14,  2,  0,  6, 10, 13, 15,  3,  5,  8 },
		{ 2,  1, 14,  7,  4, 10,  8, 13, 15, 12,  9,  0,  3,  5,  6, 11 },
	},
};
__constant__ int d_Pbox[32] = {
	16,  7, 20, 21, 29, 12, 28, 17,
	1, 15, 23, 26,  5, 18, 31, 10,
	2,  8, 24, 14, 32, 27,  3,  9,
	19, 13, 30,  6, 22, 11,  4, 25
};

/*
 * Generates 16 keys (saved in generating order when encrypting or reverse order when decrypting) and saves them into keys[] array
 */
__device__ __host__ void generate_keys(uint64_t basekey, bool reverse, const int PC1[], const int PC2[], const int Rotations[], uint64_t keys[])
{
	uint64_t first = 0;
	for (int i = 0; i < 56; i++)
	{
		if (basekey & ((uint64_t)1 << (63 - (PC1[i] - 1))))
			first += ((uint64_t)1 << 63 - i);
	}

	uint64_t d[17];
	uint64_t c[17];

	const uint64_t mask = 0b0000000000000000000000000000111111111111111111111111111100000000;
	d[0] = (first & mask) << 28; //right half
	c[0] = ((first >> 28) & mask) << 28; //left half

	for (int i = 1; i <= 16; i++)
	{
		int shifts = Rotations[i - 1];
		c[i] = c[i - 1] << shifts;
		d[i] = d[i - 1] << shifts;

		if (c[i - 1] & (uint64_t)1 << 63)
			c[i] += (uint64_t)1 << 35 + shifts;
		if (shifts == 2)
			if (c[i - 1] & (uint64_t)1 << 62)
				c[i] += (uint64_t)1 << 36;

		if (d[i - 1] & (uint64_t)1 << 63)
			d[i] += (uint64_t)1 << 35 + shifts;
		if (shifts == 2)
			if (d[i - 1] & (uint64_t)1 << 62)
				d[i] += (uint64_t)1 << 36;


		keys[i] = c[i] | (d[i] >> 28);
		uint64_t tmp = 0;
		for (int j = 0; j < 48; j++)
		{
			if (keys[i] & ((uint64_t)1 << (63 - (PC2[j] - 1))))
				tmp += ((uint64_t)1 << 63 - j);
		}
		keys[i] = tmp;
	}

	if (reverse)
	{
		for (int i = 1; i <= 8; i++)
		{
			uint64_t tmp = keys[i];
			keys[i] = keys[17 - i];
			keys[17 - i] = tmp;
		}
	}
}

/*
 * Prints all v bits from start to end given by arguments
 */
__device__ __host__ void printbits(uint64_t v, int start = 0, int end = 64)
{
	for (int ii = start; ii < end; ii++)
	{
		if (((v << ii) & FIRSTBIT) == (uint64_t)0)
			printf("0");
		else
			printf("1");
	}
	printf("\n");
}

/*
 * Permutates block using initial permutation matrix or final permutation matrix (determined by the second argument)
 */
__device__ __host__ uint64_t permutate_block(uint64_t block, bool initial, const int InitialPermutation[], const int FinalPermutation[])
{
	uint64_t permutation = 0;
	for (int i = 0; i < 64; i++)
	{
		if (initial)
		{
			if (block & ((uint64_t)1 << (63 - (InitialPermutation[i] - 1))))
				permutation += ((uint64_t)1 << 63 - i);
		}
		else if (block & ((uint64_t)1 << (63 - (FinalPermutation[i] - 1))))
			permutation += ((uint64_t)1 << 63 - i);
	}

	return permutation;
}

/*
 * Expands block using Expansion matrix given by the argument
 */
__device__ __host__ uint64_t expand(uint64_t val, const int DesExpansion[])
{
	uint64_t res = 0;
	for (int i = 0; i < 48; i++)
	{
		if (val & ((uint64_t)1 << (63 - (DesExpansion[i] - 1))))
			res += ((uint64_t)1 << 63 - i);
	}
	return res;
}

__device__ __host__ uint64_t calculate_sboxes(uint64_t val, const int DesSbox[8][4][16])
{
	uint64_t mask = 0b1111110000000000000000000000000000000000000000000000000000000000;
	uint64_t middle_bits = 0b0000000000000000000000000000000000000000000000000000000000011110;
	uint64_t ret = 0;
	for (int i = 0; i < 8; i++)
	{
		uint64_t current = (val & (mask >> (6 * i))) >> (64 - 6 * (i + 1));
		int column = (current & middle_bits) >> 1;
		int row = ((current & (1 << 5)) >> 4) + (current & 1);
		uint64_t val = DesSbox[i][row][column];
		ret += val << (60 - 4 * i);
	}
	return ret;
}

/*
 * Calculates block encryption/decryption
 */
__device__ __host__ uint64_t jechanka(uint64_t permutated, uint64_t keys[], const int PC1[], const int Rotations[], const int PC2[], 
									  const int InitialPermutation[], const int FinalPermutation[], const int DesExpansion[], 
									  const int Sbox[8][4][16], const int Pbox[])
{
	uint64_t l[17], r[17];
	uint64_t mask = 0b1111111111111111111111111111111100000000000000000000000000000000;
	l[0] = permutated & mask;
	r[0] = (permutated << 32) & mask;

	for (int i = 1; i <= 16; i++)
	{
		l[i] = r[i - 1];
		uint64_t v = calculate_sboxes(keys[i] ^ expand(r[i - 1], DesExpansion), Sbox);
		uint64_t res = 0;
		for (int j = 0; j < 32; j++)
		{
			if (v & ((uint64_t)1 << (63 - (Pbox[j] - 1))))
				res += ((uint64_t)1 << 63 - j);
		}
		r[i] = l[i - 1] ^ res;
	}

	return permutate_block(r[16] + (l[16] >> 32), false, InitialPermutation, FinalPermutation);
}

/*
 * Main DES function - decrypts or encrypts whole message (length given by MSGLEN constant)
 */
__device__ __host__ void DES(uint64_t encryptedMessage[], uint64_t decryptedMessage[], uint64_t key, const int PC1[], const int Rotations[],
							 const int PC2[], const int InitialPermutation[], const int FinalPermutation[], const int DesExpansion[],
							 const int Sbox[8][4][16], const int Pbox[], bool encrypt)
{
	uint64_t keys[17];
	generate_keys(key, !encrypt, PC1, PC2, Rotations, keys);
	for (int i = 0; i < MSGLEN; i++)
	{
		if (encrypt)
			encryptedMessage[i] = jechanka(permutate_block(decryptedMessage[i], true, InitialPermutation, FinalPermutation), keys, PC1, Rotations,
				PC2, InitialPermutation, FinalPermutation, DesExpansion, Sbox, Pbox);
		else
			decryptedMessage[i] = jechanka(permutate_block(encryptedMessage[i], true, InitialPermutation, FinalPermutation), keys, PC1, Rotations,
				PC2, InitialPermutation, FinalPermutation, DesExpansion, Sbox, Pbox);

	}
}

__global__ void worker_thread(const uint64_t message[], uint64_t encrypted[], uint64_t decrypted[], int known_zeros)
{
	uint64_t threadId = blockIdx.x * BLOCK_SIZE + threadIdx.x;
	uint64_t mask = 0b0000000000000000000000000000000000000000000000000000000001111111;
	uint64_t suffix = ((threadId & (mask << 14)) << 3) | ((threadId & (mask << 7)) << 2) | ((threadId & mask) << 1);
	uint64_t current_key = 0;
	uint64_t max = (uint64_t)1 << (35 - (known_zeros - known_zeros / 8));
	uint64_t current_message[MSGLEN];
	bool go;

	for (uint64_t i = 0; i < max && work == 1; i++)
	{
		current_key = (((i & (mask << 28)) << 5) | (((i & (mask << 21)) << 4) | ((i & (mask << 14)) << 3) | ((i & (mask << 7)) << 2) | ((i & mask) << 1)) << 24) | suffix;
		DES(encrypted, current_message, current_key, d_PC1, d_Rotations, d_PC2, d_InitialPermutation, d_FinalPermutation, d_DesExpansion, d_DesSbox, d_Pbox, false);

		go = true;
		for (int j = 0; j < MSGLEN; j++) 
		{
			if (current_message[j] != message[j]) 
			{
				go = false;
				break;
			}
		}

		if (go) 
		{
			for (int j = 0; j < MSGLEN; j++) 
			{
				decrypted[j] = current_message[j];
			}
			work = 0;
		}
	}
}

/*
 * Decrypts message using brute-force method and CUDA
 */
hipError_t CudaDES(uint64_t plaintext[], uint64_t encrypted[], uint64_t decrypted[], uint64_t key)
{
	hipError_t cudaStatus;

	uint64_t *d_plain, *d_enc, *d_dec;

	cudaStatus = hipMalloc(&d_plain, MSGLEN * sizeof(uint64_t));
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc(&d_enc, MSGLEN * sizeof(uint64_t));
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc(&d_dec, MSGLEN * sizeof(uint64_t));
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMemcpy(d_plain, plaintext, MSGLEN * sizeof(uint64_t), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	cudaStatus = hipMemcpy(d_enc, encrypted, MSGLEN * sizeof(uint64_t), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	worker_thread << < BLOCKS, BLOCK_SIZE >> > (d_plain, d_enc, d_dec, KNOWN_ZEROS);
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "worker thread failed!");
		goto Error;
	}

	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipDeviceSynchronize failed!");
		goto Error;
	}

	cudaStatus = hipMemcpy(decrypted, d_dec, MSGLEN * sizeof(uint64_t), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

Error:
	hipFree(d_plain);
	hipFree(d_enc);
	hipFree(d_dec);

	return cudaStatus;
}

/*
 * Prints message in binary format
 */
void printmsg(uint64_t msg[])
{
	for (int i = 0; i < MSGLEN; i++)
		printbits(msg[i]);
}

/*
 * Compares original message with message decrypted with CUDA/CPU
 */
bool proper_decipher(uint64_t msg[], uint64_t decrypted[])
{
	for (int i = 0; i < MSGLEN; i++)
	{
		if (msg[i] != decrypted[i])
		{
			return false;
		}
	}
	return true;
}

void cpuDES(uint64_t plaintext[], uint64_t encrypted[], uint64_t decrypted[], uint64_t key, int known_zeros)
{
	int cpu_work = 1;
	for (uint64_t threadId = 0; threadId < (uint64_t)2 << 21 && cpu_work == 1; threadId++)
	{
		uint64_t mask = 0b0000000000000000000000000000000000000000000000000000000001111111;
		uint64_t suffix = ((threadId & (mask << 14)) << 3) | ((threadId & (mask << 7)) << 2) | ((threadId & mask) << 1);
		uint64_t current_key = 0;
		uint64_t max = (uint64_t)1 << (35 - (known_zeros - known_zeros / 8));
		uint64_t current_message[MSGLEN];
		bool go;

		for (uint64_t i = 0; i < max && cpu_work == 1; i++)
		{
			current_key = (((i & (mask << 28)) << 5) | (((i & (mask << 21)) << 4) | ((i & (mask << 14)) << 3) | ((i & (mask << 7)) << 2) | ((i & mask) << 1)) << 24) | suffix;
			DES(encrypted, current_message, current_key, PC1, Rotations, PC2, InitialPermutation, FinalPermutation, DesExpansion, DesSbox, Pbox, false);

			go = true;
			for (int j = 0; j < MSGLEN; j++)
			{
				if (current_message[j] != plaintext[j])
				{
					go = false;
					break;
				}
			}

			if (go)
			{
				for (int j = 0; j < MSGLEN; j++)
				{
					decrypted[j] = current_message[j];
				}
				cpu_work = 0;
			}
		}
	}
}

int main()
{
	uint64_t key = 2;
	uint64_t msg[1] = { 0b00000000000100100011010001010110011110001001101010111100110111101111 };
	uint64_t decrypted[1];
	uint64_t encrypted[1];
	bool success;
	printf("Plain text:\n");
	printmsg(msg);

	DES(encrypted, msg, key, PC1, Rotations, PC2, InitialPermutation, FinalPermutation, DesExpansion, DesSbox, Pbox, true);

	printf("Encrypted:\n");
	printmsg(encrypted);

	clock_t begin, end;
	double elapsed_secs;

/* --------------------------------------- CUDA ----------------------------------------------------- */

	//begin = clock();
	//printf("Starting GPU DES cracking for %d known leading zeros...\n", KNOWN_ZEROS);
	//hipError_t cudaStatus = CudaDES(msg, encrypted, decrypted, key);
	//end = clock();
	//elapsed_secs = double(end - begin) / CLOCKS_PER_SEC;
	//printf("Finished GPU DES cracking for known leading zeros count = %d.\nTime elapsed: %fs.\n", KNOWN_ZEROS, elapsed_secs);
	//if (cudaStatus != hipSuccess)
	//{
	//	printf("Cos sie, cos sie popsulo...\n");
	//}
	//else
	//{
	//	printf("Decrypted:\n");
	//	printmsg(decrypted);
	//	success = proper_decipher(msg, decrypted);
	//	printf(success ? "SUCCESS\n" : "FAILURE\n");
	//}

/* --------------------------------------- CPU ----------------------------------------------------- */
	
	begin = clock();
	printf("Starting CPU DES cracking for %d known leading zeros...\n", KNOWN_ZEROS);
	cpuDES(msg, encrypted, decrypted, key, KNOWN_ZEROS);
	end = clock();
	elapsed_secs = double(end - begin) / CLOCKS_PER_SEC;
	printf("Finished CPU DES cracking for known leading zeros count = %d.\nTime elapsed: %fs.\n", KNOWN_ZEROS, elapsed_secs);
	
	printf("Decrypted:\n");
	printmsg(decrypted);
	success = proper_decipher(msg, decrypted);
	printf(success ? "SUCCESS\n" : "FAILURE\n");
	
/* --------------------------------------- DEBUG ----------------------------------------------------- */

	DES(encrypted, decrypted, key, PC1, Rotations, PC2, InitialPermutation, FinalPermutation, DesExpansion, DesSbox, Pbox, false);
	printf("Decrypted with proper key:\n");
	printmsg(decrypted);
	success = proper_decipher(msg, decrypted);
	printf(success ? "SUCCESS\n" : "FAILURE\n");

	printf("Press any key to exit program\n");
	getchar();
	return 0;
}