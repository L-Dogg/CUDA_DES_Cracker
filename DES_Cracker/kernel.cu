#include "hip/hip_runtime.h"

#include <stdio.h>
#include <cstdint>
#include <cstdlib>
#include <ctime>

#define FIRSTBIT 0x8000000000000000

const int PC1[56] = {
	57, 49, 41, 33, 25, 17,  9,
	1, 58, 50, 42, 34, 26, 18,
	10,  2, 59, 51, 43, 35, 27,
	19, 11,  3, 60, 52, 44, 36,
	63, 55, 47, 39, 31, 23, 15,
	7, 62, 54, 46, 38, 30, 22,
	14,  6, 61, 53, 45, 37, 29,
	21, 13,  5, 28, 20, 12,  4
};
const int Rotations[16] = {
	1, 1, 2, 2, 2, 2, 2, 2, 1, 2, 2, 2, 2, 2, 2, 1
};
const int PC2[48] = {
	14, 17, 11, 24,  1,  5,
	3, 28, 15,  6, 21, 10,
	23, 19, 12,  4, 26,  8,
	16,  7, 27, 20, 13,  2,
	41, 52, 31, 37, 47, 55,
	30, 40, 51, 45, 33, 48,
	44, 49, 39, 56, 34, 53,
	46, 42, 50, 36, 29, 32
};

// Permutation tables

const int InitialPermutation[64] = {
	58, 50, 42, 34, 26, 18, 10,  2,
	60, 52, 44, 36, 28, 20, 12,  4,
	62, 54, 46, 38, 30, 22, 14,  6,
	64, 56, 48, 40, 32, 24, 16,  8,
	57, 49, 41, 33, 25, 17,  9,  1,
	59, 51, 43, 35, 27, 19, 11,  3,
	61, 53, 45, 37, 29, 21, 13,  5,
	63, 55, 47, 39, 31, 23, 15,  7
};
const int FinalPermutation[64] = {
	40,  8, 48, 16, 56, 24, 64, 32,
	39,  7, 47, 15, 55, 23, 63, 31,
	38,  6, 46, 14, 54, 22, 62, 30,
	37,  5, 45, 13, 53, 21, 61, 29,
	36,  4, 44, 12, 52, 20, 60, 28,
	35,  3, 43, 11, 51, 19, 59, 27,
	34,  2, 42, 10, 50, 18, 58, 26,
	33,  1, 41,  9, 49, 17, 57, 25
};

// Rounds tables

const int DesExpansion[48] = {
	32,  1,  2,  3,  4,  5,  4,  5,
	6,  7,  8,  9,  8,  9, 10, 11,
	12, 13, 12, 13, 14, 15, 16, 17,
	16, 17, 18, 19, 20, 21, 20, 21,
	22, 23, 24, 25, 24, 25, 26, 27,
	28, 29, 28, 29, 30, 31, 32,  1
};

const int DesSbox[8][4][16] = {
	{
		{ 14,  4, 13,  1,  2, 15, 11,  8,  3, 10,  6, 12,  5,  9,  0,  7 },
		{ 0, 15,  7,  4, 14,  2, 13,  1, 10,  6, 12, 11,  9,  5,  3,  8 },
		{ 4,  1, 14,  8, 13,  6,  2, 11, 15, 12,  9,  7,  3, 10,  5,  0 },
		{ 15, 12,  8,  2,  4,  9,  1,  7,  5, 11,  3, 14, 10,  0,  6, 13 },
	},

	{
		{ 15,  1,  8, 14,  6, 11,  3,  4,  9,  7,  2, 13, 12,  0,  5, 10 },
		{ 3, 13,  4,  7, 15,  2,  8, 14, 12,  0,  1, 10,  6,  9, 11,  5 },
		{ 0, 14,  7, 11, 10,  4, 13,  1,  5,  8, 12,  6,  9,  3,  2, 15 },
		{ 13,  8, 10,  1,  3, 15,  4,  2, 11,  6,  7, 12,  0,  5, 14,  9 },
	},

	{
		{ 10,  0,  9, 14,  6,  3, 15,  5,  1, 13, 12,  7, 11,  4,  2,  8 },
		{ 13,  7,  0,  9,  3,  4,  6, 10,  2,  8,  5, 14, 12, 11, 15,  1 },
		{ 13,  6,  4,  9,  8, 15,  3,  0, 11,  1,  2, 12,  5, 10, 14,  7 },
		{ 1, 10, 13,  0,  6,  9,  8,  7,  4, 15, 14,  3, 11,  5,  2, 12 },
	},

	{
		{ 7, 13, 14,  3,  0,  6,  9, 10,  1,  2,  8,  5, 11, 12,  4, 15 },
		{ 13,  8, 11,  5,  6, 15,  0,  3,  4,  7,  2, 12,  1, 10, 14,  9 },
		{ 10,  6,  9,  0, 12, 11,  7, 13, 15,  1,  3, 14,  5,  2,  8,  4 },
		{ 3, 15,  0,  6, 10,  1, 13,  8,  9,  4,  5, 11, 12,  7,  2, 14 },
	},

	{
		{ 2, 12,  4,  1,  7, 10, 11,  6,  8,  5,  3, 15, 13,  0, 14,  9 },
		{ 14, 11,  2, 12,  4,  7, 13,  1,  5,  0, 15, 10,  3,  9,  8,  6 },
		{ 4,  2,  1, 11, 10, 13,  7,  8, 15,  9, 12,  5,  6,  3,  0, 14 },
		{ 11,  8, 12,  7,  1, 14,  2, 13,  6, 15,  0,  9, 10,  4,  5,  3 },
	},

	{
		{ 12,  1, 10, 15,  9,  2,  6,  8,  0, 13,  3,  4, 14,  7,  5, 11 },
		{ 10, 15,  4,  2,  7, 12,  9,  5,  6,  1, 13, 14,  0, 11,  3,  8 },
		{ 9, 14, 15,  5,  2,  8, 12,  3,  7,  0,  4, 10,  1, 13, 11,  6 },
		{ 4,  3,  2, 12,  9,  5, 15, 10, 11, 14,  1,  7,  6,  0,  8, 13 },
	},

	{
		{ 4, 11,  2, 14, 15,  0,  8, 13,  3, 12,  9,  7,  5, 10,  6,  1 },
		{ 13,  0, 11,  7,  4,  9,  1, 10, 14,  3,  5, 12,  2, 15,  8,  6 },
		{ 1,  4, 11, 13, 12,  3,  7, 14, 10, 15,  6,  8,  0,  5,  9,  2 },
		{ 6, 11, 13,  8,  1,  4, 10,  7,  9,  5,  0, 15, 14,  2,  3, 12 },
	},

	{
		{ 13,  2,  8,  4,  6, 15, 11,  1, 10,  9,  3, 14,  5,  0, 12,  7 },
		{ 1, 15, 13,  8, 10,  3,  7,  4, 12,  5,  6, 11,  0, 14,  9,  2 },
		{ 7, 11,  4,  1,  9, 12, 14,  2,  0,  6, 10, 13, 15,  3,  5,  8 },
		{ 2,  1, 14,  7,  4, 10,  8, 13, 15, 12,  9,  0,  3,  5,  6, 11 },
	},
};

const int Pbox[32] = {
	16,  7, 20, 21, 29, 12, 28, 17,
	1, 15, 23, 26,  5, 18, 31, 10,
	2,  8, 24, 14, 32, 27,  3,  9,
	19, 13, 30,  6, 22, 11,  4, 25
};

void printbits(uint64_t v, int start = 0, int end = 64);
uint64_t* generate_keys(uint64_t basekey);
uint64_t permutate_block(uint64_t block, bool initial);
uint64_t jechanka(uint64_t permutated, uint64_t* keys);
uint64_t expand(uint64_t val);
uint64_t calculate_sboxes(uint64_t val);

int main()
{
	uint64_t key = 0b0001001100110100010101110111100110011011101111001101111111110001;
	uint64_t msg = 0b0000000100100011010001010110011110001001101010111100110111101111;

	uint64_t* keys = generate_keys(key);
	jechanka(permutate_block(msg, true), keys);
    return 0;
}

uint64_t* generate_keys(uint64_t basekey)
{
	uint64_t* keys = (uint64_t *)malloc(16 * sizeof(uint64_t));
	uint64_t first = 0;
	for (int i = 0; i < 56; i++)
	{
		if (basekey & ((uint64_t) 1 << (63 - (PC1[i] - 1))))
			first += ((uint64_t) 1 << 63 - i);
	}

	uint64_t d[17];
	uint64_t c[17];

	const uint64_t mask = 0b000000000000000000000000000111111111111111111111111111110000000;
	d[0] = (first & mask) << 28; //right half
	c[0] = ((first >> 28) & mask) << 28; //left half
	
	for (int i = 1; i <= 16; i++)
	{
		int shifts = Rotations[i-1];
		c[i] = c[i - 1] << shifts;
		d[i] = d[i - 1] << shifts;

		if (c[i - 1] & (uint64_t)1 << 63)
			c[i] += (uint64_t)1 << 35 + shifts;
		if (shifts == 2)
			if (c[i - 1] & (uint64_t)1 << 62)
				c[i] += (uint64_t)1 << 36;

		if (d[i - 1] & (uint64_t)1 << 63)
			d[i] += (uint64_t)1 << 35 + shifts;
		if (shifts == 2)
			if (d[i - 1] & (uint64_t)1 << 62)
				d[i] += (uint64_t)1 << 36;

		
		keys[i] = c[i] | (d[i] >> 28);
		uint64_t tmp = 0;
		for (int j = 0; j < 48; j++)
		{
			if (keys[i] & ((uint64_t)1 << (63 - (PC2[j] - 1))))
				tmp += ((uint64_t)1 << 63 - j);
		}

		keys[i] = tmp;
	}
	return keys;
}

void printbits(uint64_t v, int start, int end)
{
	for (int ii = start; ii < end; ii++)
	{
		if (((v << ii) & FIRSTBIT) == (uint64_t)0)
			printf("0");
		else
			printf("1");
	}
	printf("\n");
}

uint64_t permutate_block(uint64_t block, bool initial)
{
	uint64_t permutation = 0;
	for (int i = 0; i < 64; i++)
	{
		if (initial)
		{
			if (block & ((uint64_t)1 << (63 - (InitialPermutation[i] - 1))))
				permutation += ((uint64_t)1 << 63 - i);
		}
		else if (block & ((uint64_t)1 << (63 - (FinalPermutation[i] - 1))))
			permutation += ((uint64_t)1 << 63 - i);	
	}

	return permutation;
}

uint64_t jechanka(uint64_t permutated, uint64_t* keys)
{
	uint64_t l[17], r[17];
	uint64_t mask = 0b1111111111111111111111111111111100000000000000000000000000000000;
	l[0] = permutated & mask;
	r[0] = (permutated << 32) & mask;

	for(int i = 1; i <= 16; i++)
	{
		l[i] = r[i - 1];
		uint64_t v = calculate_sboxes(keys[i] ^ expand(r[i - 1]));
		uint64_t res = 0;
		for (int j = 0; i < 32; i++)
		{
			if (v & ((uint64_t)1 << (63 - (Pbox[j] - 1))))
				res += ((uint64_t)1 << 63 - j);
		}
		r[i] = l[i - 1] ^ res;
	}

	return permutate_block(r[16] | (l[16] >> 32), false);
}

uint64_t expand(uint64_t val)
{
	uint64_t res = 0;
	for (int i = 0; i < 48; i++)
	{
		if (val & ((uint64_t)1 << (63 - (DesExpansion[i] - 1))))
			res += ((uint64_t)1 << 63 - i);
	}
	return res;
}

uint64_t calculate_sboxes(uint64_t val)
{
	uint64_t mask = 0b1111110000000000000000000000000000000000000000000000000000000000;
	uint64_t middle_bits = 0b0000000000000000000000000000000000000000000000000000000000011110;
	uint64_t ret = 0;
	for(int i = 0; i < 8; i++)
	{
		uint64_t current = (val & (mask >> (6 * i))) >> (64 - 6 * (i + 1));
		int column = (current & middle_bits) >> 1;
		int row = ((current & (1 << 5)) >> 4) + (current & 1);
		uint64_t val = DesSbox[i][row][column];
		ret += val << (60 - 4 * i);
	}
	return ret;
}
